#include <mvgmm.h>
#include <thrust/device_vector.h>

int main(int argc, char *argv[]) {

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;

    int K = 4;
    int N = 32768;
    int D = 3;

    // Set values for K mean vectors and K covariance matrices

    float h_weights[K] = {0.2, 0.4, 0.3, 0.1};
    float h_means[K][D] = {{0, 0, 0}, {-10, 7, 3}, {5, 1, 12}, {1, -5, -2}};
    float h_covariances[K][D*D] = {{1, 0, 0, 0, 1, 0, 0, 0, 1},
                                   {2, 0.9, 0.1, 0.9, 1, 1.5, 0.1, 1.5, 0},
                                   {1, -0.9, 1.2, -0.9, 4, 0.4, 1.2, 0.4, 2},
                                   {1.5, 0.7, -1.1, 0.7, 2.2, 0, -1.1, 0, 0.7}};

    // Create handles

    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);

    // Copy to device

    thrust::device_vector<float> d_weights(K);
    thrust::device_vector<float> d_means(K * D);
    thrust::device_vector<float> d_covariances(K * D * D);
    thrust::device_vector<float> d_randomValues(D * N);

    for (int k = 0; k < K; k++) {
        hipMemcpy(thrust::raw_pointer_cast(d_weights.data() + k), &h_weights[k],
                   sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(thrust::raw_pointer_cast(d_means.data() + k * D), &h_means[k * D],
                   sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(thrust::raw_pointer_cast(d_covariances.data() + k * D * D), &h_covariances[k * D * D],
                   sizeof(float), hipMemcpyHostToDevice);
    }

    rmvgmm(cublasHandle, cusolverHandle, N, D, K, thrust::raw_pointer_cast(d_weights.data()),
           thrust::raw_pointer_cast(d_means.data()), thrust::raw_pointer_cast(d_covariances.data()),
           thrust::raw_pointer_cast(d_randomValues.data()));

    // Cleanup

    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);

    return 0;
}
